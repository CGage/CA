
#include "hip/hip_runtime.h"


#include "iostream"
#include "stdlib.h"
#include <thread>         // std::this_thread::sleep_for
#include <chrono>         // std::chrono::seconds
#include "time.h"
#include <ctime>
#include "fstream"



using namespace std;

int getPos(int m, int n, const int width) {
	return m* width + n;
}

void printCells(int* cells, int const height, int const width) {
	for (int i = 0; i < height + 2; i++) {
		for (int j = 0; j < width + 2; j++) { 
			if (cells[getPos(i, j, width)] == 1) {
				cout << "O" << " ";
			}
			else {
				cout << "-" << " ";
			}
		}
		cout << endl;
	}
	cout << endl;
	std::this_thread::sleep_for(std::chrono::milliseconds(100));
	system("cls");
}

void populateArray(int* cellArray, int arraySize) {
	for (int i = 0; i < arraySize; i++) { 
		cellArray[i] = rand() % 2;
	}
}

__device__ int getX(int i, int width) {
	return i % width;
}

__device__ int getY(int i, int width) {
	return i / width;
}

__device__ int getI(int m, int n, int width) {
	return m * width + n;
}


//Gets the neigbour cells via von Neuman Neigbourhood 
__device__ int getNeigbours(int m, int n, int* cells, int width, int height) {
	int neigbours = 0;
	for (int i = m - 1; i <= m + 1; i++) {
		for (int j = n - 1; j <= n + 1; j++) {
			if (i >= 0 && i < height && j >= 0 && j < width) {
				neigbours += cells[getI(i, j, width)];
			}
			else {
				neigbours += cells[getI((i + height) % height, (j + width) % width, width)];
			}
		}
	}
	return neigbours;
}

// rules that determines the state of the cell
__device__ int rules(int neigbours, int state) {
	int n = neigbours - state;
	if (state == 1) {
		if (n > 1 && n < 4) {
			return 1;
		}
		else {
			return 0;
		}
	}
	else {
		if (n == 3){
			return 1;
		}
		return 0;
	}

}

// creates the new state of the world
__global__ void evolve(int* cells, const int height, const int width, const int arraySize, const int cellsPerThread) {
	extern __shared__ int sharedCells[];
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	for (int k = i * cellsPerThread; k < ((i + 1) * cellsPerThread); k++) {		
		sharedCells[k] = cells[k];
		int x, y, neigbours;
		x = getX(k, width);
		y = getY(k, width);
		neigbours = getNeigbours(y, x, sharedCells, width, height);
		cells[k] = rules(neigbours, sharedCells[getI(y, x, width)]);
		__syncthreads();
	}

	
}

// Runs the simulation
int main() {
	srand(1);
	const int height = 100, width = 100, arraySize = 10000, timeSteps = 10000, cellsPerThread = 10, gridSize = 10;
	char b;
	int* cells; // CPU
	int* cellsDev; // GPU

	cells = (int*)malloc(sizeof(int)*arraySize); // creating arrays
	populateArray(cells, arraySize);

	hipMalloc((void**)&cellsDev, sizeof(float)*arraySize); // creating space on gpu

	hipMemcpy(cellsDev, cells, sizeof(int)*arraySize, hipMemcpyHostToDevice); // copying arrays to gpu
	clock_t begin = clock();
	for (int i = 1; i < timeSteps; i++) {
		evolve <<<gridSize, arraySize / cellsPerThread / gridSize >>>(cellsDev, height, width, arraySize, cellsPerThread); // running evolution iteration	
	}
	clock_t end = clock();
	
	hipMemcpy(cells, cellsDev, sizeof(int)*arraySize, hipMemcpyDeviceToHost); // copying cells back from gpu to cpu	
	hipFree(cellsDev);

	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	cout << elapsed_secs;

	ofstream myfile;
	myfile.open("para4.txt");
	for (int i = 0; i < arraySize; i++) {
		myfile << cells[i] << endl;
	}
	free(cells);
	myfile.close();
	cin >> b;
	return 0;
}
